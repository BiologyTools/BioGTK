
#include <hip/hip_runtime.h>
__global__ void copyTileToCanvas(unsigned char* canvas, int canvasWidth, int canvasHeight,
    unsigned char* tile, int tileWidth, int tileHeight,
    int offsetX, int offsetY) 
{
    // Calculate the global x and y index for the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the source and destination indices
    int tileIdx = (y * tileWidth + x) * 3; // Each pixel has 3 components (RGB)
    int canvasIdx = ((y + offsetY) * canvasWidth + (x + offsetX)) * 3;
    // Copy the pixel (RGB components)
    canvas[canvasIdx] = tile[tileIdx];
    canvas[canvasIdx + 1] = tile[tileIdx + 1];
    canvas[canvasIdx + 2] = tile[tileIdx + 2];
}
