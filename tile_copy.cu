
#include <hip/hip_runtime.h>
// Struct to define the extent of a tile
struct Extent {
    double MinX;
    double MinY;
    double MaxX;
    double MaxY;
};

// Struct to hold tile data
struct TileData {
    Extent Extent;              // The extent of the tile
    unsigned char* DevTilePtr;  // Pointer to the tile data on the GPU
};

__global__ void StitchKernel(
    unsigned char* devCanvas,
    int canvasWidth,
    int canvasHeight,
    TileData* devTiles,
    int tileCount,
    double minX,
    double minY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < canvasWidth && y < canvasHeight)
    {
        for (int i = 0; i < tileCount; ++i)
        {
            Extent extent = devTiles[i].Extent;
            unsigned char* tileData = (unsigned char*)devTiles[i].DevTilePtr;

            int startX = (int)(extent.MinX - minX);
            int startY = (int)(extent.MinY - minY);
            int tileWidth = (int)(extent.MaxX - extent.MinX);
            int tileHeight = (int)(extent.MaxY - extent.MinY);

            if (x >= startX && x < startX + tileWidth && y >= startY && y < startY + tileHeight)
            {
                int canvasIndex = (y * canvasWidth + x) * 3;
                int tileIndex = ((y - startY) * tileWidth + (x - startX)) * 3;

                // Avoid race conditions by ensuring only one tile writes to each pixel.
                devCanvas[canvasIndex] = tileData[tileIndex];
                devCanvas[canvasIndex + 1] = tileData[tileIndex + 1];
                devCanvas[canvasIndex + 2] = tileData[tileIndex + 2];
            }
        }
    }
}