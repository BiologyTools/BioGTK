
#include <hip/hip_runtime.h>
__global__ void copyTileToCanvas(
    unsigned char* canvas, int canvasWidth, int canvasHeight,
    unsigned char* tile, int tileWidth, int tileHeight,
    int offsetX, int offsetY,
    int canvasTileWidth, int canvasTileHeight)
{
    // Calculate the global x and y index for the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure this thread only processes pixels within the bounds of the tile and canvas section
    if (x < canvasTileWidth && y < canvasTileHeight) {
        int canvasX = x + offsetX;
        int canvasY = y + offsetY;

        // Ensure canvas indices are within bounds of the canvas extent
        if (canvasX < canvasWidth && canvasY < canvasHeight) {
            // Calculate scaling factors to map canvasTileWidth and canvasTileHeight to tile dimensions
            float scaleX = static_cast<float>(tileWidth) / static_cast<float>(canvasTileWidth);
            float scaleY = static_cast<float>(tileHeight) / static_cast<float>(canvasTileHeight);

            // Compute the corresponding tile indices
            int tileX = min(static_cast<int>(x * scaleX), tileWidth - 1);
            int tileY = min(static_cast<int>(y * scaleY), tileHeight - 1);

            // Calculate the indices for both canvas and tile in the 1D arrays
            int tileIdx = (tileY * tileWidth + tileX) * 3;   // Each pixel has 3 components (RGB)
            int canvasIdx = (canvasY * canvasWidth + canvasX) * 3;

            // Copy the pixel (RGB components) from the tile to the canvas
            canvas[canvasIdx] = tile[tileIdx];        // Blue component
            canvas[canvasIdx + 1] = tile[tileIdx + 1];    // Green component
            canvas[canvasIdx + 2] = tile[tileIdx + 2];        // Red component
        }
    }
}
